#include "hip/hip_runtime.h"
//
// Created by peterglenn on 5/12/25.
//

#include "cuda.cuh"
#include "cudaengine.cuh"
#include "cudafn.cu"

#include <hip/hip_runtime.h>


__global__ void wrapCUDAeval( bool* crit, CUDAvalms* out, CUDAextendedcontext* Cecs, const CUDAfcptr start, const uint sz )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sz)
    {
        crit[index] = true;
        out[index] = CUDAevalinternal(Cecs[index],start);
    }
}

__global__ void wrapCUDAevalcriterion( bool* crit, CUDAvalms* out, CUDAextendedcontext* Cecs, const CUDAfcptr start, const uint sz )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sz)
    {
        crit[index] = CUDAto_mtbool(CUDAevalinternal(Cecs[index],Cecs[index].CUDAfcarray[start].criterion));
        if (crit[index] == true)
            out[index] = CUDAevalinternal(Cecs[index],start);
    }
}

__global__ void wrapCUDAevalfast( bool* crit, CUDAvalms* out, CUDAextendedcontext& Cec,
    const CUDAfcptr start, const uint dimm, const uint sz )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    CUDAvdimn v;
    int i = 0;
    int r = index;
    while (i < GPUQUANTFASTDIM)
    {
        v[i] = r % dimm;
        r /= dimm;
        ++i;
    }
    if (index < sz)
    {
        CUDAextendedcontext Cecplus = Cec;
        for (int j = 0; j < Cecplus.numfastn; ++j)
            Cecplus.fastn[j] = v[j];
        crit[index] = true;
        out[index] = CUDAevalinternal(Cecplus,start);
    }
}

__global__ void wrapCUDAevalcriterionfast( bool* crit, CUDAvalms* out, CUDAextendedcontext& Cec,
    const CUDAfcptr start, const uint dimm, const uint sz )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    CUDAvdimn v;
    int i = 0;
    int r = index;
    while (i < GPUQUANTFASTDIM)
    {
        v[i] = r % dimm;
        r /= dimm;
        ++i;
    }
    if (index < sz)
    {
        CUDAextendedcontext Cecplus = Cec;
        for (int j = 0; j < Cecplus.numfastn; ++j)
            Cecplus.fastn[j] = v[j];
        crit[index] = CUDAto_mtbool(CUDAevalinternal(Cecplus,Cec.CUDAfcarray[start].criterion));
        if (crit[index] == true)
            out[index] = CUDAevalinternal(Cecplus,start);
    }
}

void CUDAevalwithcriterionfast( bool* crit, CUDAvalms* out, CUDAextendedcontext& Cec, const CUDAfcptr start,
    const uint dimm, const uint sz )
{
    auto starttime = std::chrono::high_resolution_clock::now();

    CUDAextendedcontext h_Cec = Cec;
    CUDAextendedcontext* d_Cec;

    hipMalloc((void**)&h_Cec.CUDAfcarray, Cec.CUDAfcarraysize*sizeof(CUDAfc));
    hipMalloc((void**)&h_Cec.namedvararray, Cec.namedvararraysize*sizeof(CUDAnamedvariable));
    hipMalloc((void**)&h_Cec.CUDAvalsarray, Cec.CUDAvalsarraysize);
    hipMalloc((void**)&h_Cec.CUDAcontext, Cec.CUDAcontextsize*sizeof(CUDAnamedvariable));
    hipMalloc((void**)&h_Cec.CUDAliteralarray, Cec.CUDAliteralarraysize*sizeof(CUDAliteral));
    hipMalloc((void**)&h_Cec.fastn,sizeof(CUDAvdimn));

    hipMemcpy(h_Cec.CUDAfcarray, Cec.CUDAfcarray,Cec.CUDAfcarraysize*sizeof(CUDAfc),hipMemcpyHostToDevice);
    hipMemcpy(h_Cec.namedvararray, Cec.namedvararray,Cec.namedvararraysize*sizeof(CUDAnamedvariable),hipMemcpyHostToDevice);
    hipMemcpy(h_Cec.CUDAvalsarray, Cec.CUDAvalsarray,Cec.CUDAvalsarraysize,hipMemcpyHostToDevice);
    hipMemcpy(h_Cec.CUDAcontext, Cec.CUDAcontext,Cec.CUDAcontextsize*sizeof(CUDAnamedvariable),hipMemcpyHostToDevice);
    hipMemcpy(h_Cec.CUDAliteralarray, Cec.CUDAliteralarray,Cec.CUDAliteralarraysize*sizeof(CUDAliteral),hipMemcpyHostToDevice);

    hipMalloc((void**)&d_Cec,sizeof(CUDAextendedcontext));

    hipMemcpy(d_Cec, &h_Cec,sizeof(CUDAextendedcontext),hipMemcpyHostToDevice);

    CUDAvalms *d_out;
    bool *d_crit;

    hipMalloc((void**)&d_out, sz*sizeof(CUDAvalms));
    hipMalloc((void**)&d_crit, sz*sizeof(bool));

    int blockSize = 256;
    int numBlocks = (sz + blockSize - 1) / blockSize;


#ifdef CUDADEBUG2
    auto starttime2 = std::chrono::high_resolution_clock::now();
#endif

    size_t pValue;
    hipDeviceGetLimit(&pValue,hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize,2048);
    hipDeviceGetLimit(&pValue,hipLimitStackSize);

#ifdef CUDADEBUG2

    std::cout << "hipDeviceGetLimit(hipLimitStackSize) == " << pValue << std::endl;

#endif

    populateCUDAfnarraysingle<<<numBlocks,blockSize>>>(d_Cec);

    if (sz > 0)
        if (Cec.CUDAfcarray[start].criterion >= 0)
            wrapCUDAevalcriterionfast<<<numBlocks,blockSize>>>(d_crit, d_out, *d_Cec, start, dimm, sz);
        else
            wrapCUDAevalfast<<<numBlocks,blockSize>>>(d_crit, d_out, *d_Cec, start, dimm, sz);

    hipDeviceSynchronize();
#ifdef CUDADEBUG2

    auto stoptime2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(stoptime2 - starttime2);

    std::cout << "CUDA runtime excluding hipMalloc and hipMemcpy: " << duration2.count() << " microseconds" << std::endl;
#endif
    hipMemcpy( crit, d_crit, sz * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy( out, d_out, sz * sizeof(CUDAvalms), hipMemcpyDeviceToHost);

    hipFree(h_Cec.CUDAfcarray);
    hipFree(h_Cec.namedvararray);
    hipFree(h_Cec.CUDAvalsarray);
    hipFree(h_Cec.CUDAcontext);
    hipFree(h_Cec.CUDAliteralarray);

    hipFree(d_out);
    hipFree(d_crit);

#ifdef CUDADEBUG2
    auto stoptime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stoptime - starttime);

    std::cout << "CUDA runtime including hipMalloc and hipMemcpy: " << duration.count() << " microseconds" << std::endl;
#endif


#ifdef CUDADEBUG
    for (int i = 0; i < sz; ++i)
        std::cout << "OUT: " << i << ": " << crit[i] << ".. " << out[i].v.bv << "(type " << out[i].t << ")" << "; ";
    std::cout << std::endl;
#endif


}


void CUDAevalwithcriterion( bool* crit, CUDAvalms* out, CUDAextendedcontext* Cecs, const CUDAfcptr start, const uint sz )
{
    auto starttime = std::chrono::high_resolution_clock::now();

    CUDAextendedcontext h_Cecs[sz];
    CUDAextendedcontext* d_Cecs;
    hipMalloc((void**)&d_Cecs,sz*sizeof(CUDAextendedcontext));

    for (int i = 0; i < sz; ++i)
    {
        h_Cecs[i] = Cecs[i];

        hipMalloc((void**)&h_Cecs[i].CUDAfcarray,Cecs[i].CUDAfcarraysize*sizeof(CUDAfc));
        hipMalloc((void**)&h_Cecs[i].namedvararray, Cecs[i].namedvararraysize*sizeof(CUDAnamedvariable));
        hipMalloc((void**)&h_Cecs[i].CUDAvalsarray,Cecs[i].CUDAvalsarraysize);
        hipMalloc((void**)&h_Cecs[i].CUDAcontext, Cecs[i].CUDAcontextsize*sizeof(CUDAnamedvariable));
        hipMalloc((void**)&h_Cecs[i].CUDAliteralarray, Cecs[i].CUDAliteralarraysize*sizeof(CUDAliteral));
        // hipMalloc((void**)&h_Cecs[i].fastn,sizeof(CUDAvdimn));

        hipMemcpy(h_Cecs[i].CUDAfcarray, Cecs[i].CUDAfcarray,Cecs[i].CUDAfcarraysize*sizeof(CUDAfc),hipMemcpyHostToDevice);
        hipMemcpy(h_Cecs[i].namedvararray, Cecs[i].namedvararray,Cecs[i].namedvararraysize*sizeof(CUDAnamedvariable),hipMemcpyHostToDevice);
        hipMemcpy(h_Cecs[i].CUDAvalsarray, Cecs[i].CUDAvalsarray,Cecs[i].CUDAvalsarraysize,hipMemcpyHostToDevice);
        hipMemcpy(h_Cecs[i].CUDAcontext, Cecs[i].CUDAcontext,Cecs[i].CUDAcontextsize*sizeof(CUDAnamedvariable),hipMemcpyHostToDevice);
        hipMemcpy(h_Cecs[i].CUDAliteralarray, Cecs[i].CUDAliteralarray,Cecs[i].CUDAliteralarraysize*sizeof(CUDAliteral),hipMemcpyHostToDevice);
        // hipMemcpy(h_Cecs[i].fastn, Cecs[i].fastn,sizeof(CUDAvdimn),hipMemcpyHostToDevice);

#ifdef CUDADEBUG
        for (int j = 0; j < Cecs[i].CUDAfcarraysize; ++j)
        {
            auto tempfc = Cecs[i].CUDAfcarray[j];
            std::cout << "CUDAeval: " << j << " (" << int(tempfc.fo) << "): " << tempfc.criterion << ", " << tempfc.fcleft << ", " << tempfc.fcright << std::endl;
        }
#endif
    }

    hipMemcpy(d_Cecs, h_Cecs,sz*sizeof(CUDAextendedcontext),hipMemcpyHostToDevice);

    CUDAvalms *d_out;
    bool *d_crit;

    hipMalloc((void**)&d_out, sz*sizeof(CUDAvalms));
    hipMalloc((void**)&d_crit, sz*sizeof(bool));

    int blockSize = 1024;
    int numBlocks = (sz + blockSize - 1) / blockSize;


#ifdef CUDADEBUG2
    auto starttime2 = std::chrono::high_resolution_clock::now();
#endif

    size_t pValue;
    hipDeviceGetLimit(&pValue,hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize,2048);
    hipDeviceGetLimit(&pValue,hipLimitStackSize);

#ifdef CUDADEBUG2

    std::cout << "hipDeviceGetLimit(hipLimitStackSize) == " << pValue << std::endl;

#endif


    if (sz > 0)
    {
        populateCUDAfnarray<<<numBlocks,blockSize>>>(d_Cecs,sz);
        if (Cecs[0].CUDAfcarray[start].criterion >= 0)
            wrapCUDAevalcriterion<<<numBlocks,blockSize>>>(d_crit, d_out, d_Cecs, start, sz);
        else
            wrapCUDAeval<<<numBlocks,blockSize>>>(d_crit, d_out, d_Cecs, start, sz);
    }

    hipDeviceSynchronize();
#ifdef CUDADEBUG2

    auto stoptime2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(stoptime2 - starttime2);

    std::cout << "CUDA runtime excluding hipMalloc and hipMemcpy: " << duration2.count() << " microseconds" << std::endl;
#endif
    hipMemcpy( out, d_out, sz * sizeof(CUDAvalms), hipMemcpyDeviceToHost);
    hipMemcpy( crit, d_crit, sz * sizeof(bool), hipMemcpyDeviceToHost);

    for (auto i = 0; i < sz; ++i)
    {
        hipFree(h_Cecs[i].CUDAfcarray);
        hipFree(h_Cecs[i].namedvararray);
        hipFree(h_Cecs[i].CUDAvalsarray);
        hipFree(h_Cecs[i].CUDAcontext);
        hipFree(h_Cecs[i].CUDAliteralarray);
    }
    hipFree(d_out);
    hipFree(d_crit);
    hipFree(h_Cecs);

#ifdef CUDADEBUG2
    auto stoptime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stoptime - starttime);

    std::cout << "CUDA runtime including hipMalloc and hipMemcpy: " << duration.count() << " microseconds" << std::endl;
#endif


#ifdef CUDADEBUG
    for (int i = 0; i < sz; ++i)
        std::cout << "OUT: " << i << ": " << crit[i] << ".. " << out[i].v.iv << "(type " << out[i].t << ")" << "; ";
    std::cout << std::endl;
#endif
}

void CUDAevalwithcriterion( bool* crit, CUDAvalms* out, CUDAextendedcontext* Cecs, const uint sz )
{
    CUDAevalwithcriterion(crit, out, Cecs, Cecs[0].fctop, sz);
}

